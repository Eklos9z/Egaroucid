#include "hip/hip_runtime.h"
/*
    Egaroucid Project

    @file eval_optimizer_cuda.cu
        Evaluation Function Optimizer in CUDA
    @date 2021-2024
    @author Takuto Yamana
    @license GPL-3.0 license
*/

#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <vector>
#include <unordered_set>
#include <string>
#include <fstream>
#include <iostream>
#include <numeric>
#include <iterator>
#include <random>
#include <algorithm>
#include <time.h>
#include <chrono>
#define OPTIMIZER_INCLUDE
#include "evaluation_definition.hpp"

// train data constant
#define ADJ_MAX_N_FILES 64
#if ADJ_CELL_WEIGHT
    #define ADJ_MAX_N_DATA 1000000
#else
    #define ADJ_MAX_N_DATA 100000000
#endif
#define ADJ_MAX_N_TEST_DATA 100000

// GPU constant
#define N_THREADS_PER_BLOCK_TEST 1024
#define N_THREADS_PER_BLOCK_RESIDUAL 1024
#define N_THREADS_PER_BLOCK_NEXT_STEP 1024


// monitor constant
#define N_ERROR_MONITOR 2 // 0 for MSE, 1 for MAE
#define N_TEST_ERROR_MONITOR 2 // 0 for MSE, 1 for MAE


struct Adj_Data {
    uint16_t features[ADJ_N_FEATURES];
    float score;
};



/*
    @brief timing function

    @return time in milliseconds
*/
inline uint64_t tim(){
    return std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now().time_since_epoch()).count();
}

/*
    @brief initialize some arrays
*/
void adj_init_arr(int eval_size, float *host_eval_arr, int *host_rev_idx_arr, int *host_n_appear_arr) {
    for (int i = 0; i < eval_size; ++i) {
        host_eval_arr[i] = 0.0;
        host_n_appear_arr[i] = 0;
    }
    int strt_idx = 0;
    for (int i = 0; i < ADJ_N_EVAL; ++i) {
        for (int j = 0; j < adj_eval_sizes[i]; ++j) {
            host_rev_idx_arr[strt_idx + j] = strt_idx + adj_calc_rev_idx(i, j);
        }
        strt_idx += adj_eval_sizes[i];
    }
}

/*
    @brief import pre-calculated evaluation function
*/
void adj_import_eval(std::string file, int eval_size, float *host_eval_arr) {
    std::ifstream ifs(file);
    if (ifs.fail()) {
        std::cerr << "evaluation file " << file << " not exist" << std::endl;
        return;
    }
    std::cerr << "importing eval params " << file << std::endl;
    std::string line;
    for (int i = 0; i < eval_size; ++i){
        if (!getline(ifs, line)) {
            std::cerr << "ERROR evaluation file broken" << std::endl;
            return;
        }
        host_eval_arr[i] = stof(line);
    }
}

/*
    @brief import train data
*/
int adj_import_data(int n_files, char* files[], Adj_Data* host_train_data, int *host_rev_idx_arr, int *host_n_appear_arr) {
    int n_data = 0;
    FILE* fp;
    int16_t n_discs, score, player;
    Adj_Data data;
    float score_avg = 0.0;
    int start_idx_arr[ADJ_N_FEATURES];
    int start_idx = 0;
    for (int i = 0; i < ADJ_N_FEATURES; ++i){
        if (i > 0){
            if (adj_feature_to_eval_idx[i] > adj_feature_to_eval_idx[i - 1]){
                start_idx += adj_eval_sizes[adj_feature_to_eval_idx[i - 1]];
            }
        }
        start_idx_arr[i] = start_idx;
    }
    for (int file_idx = 0; file_idx < n_files; ++file_idx) {
        std::cerr << files[file_idx] << std::endl;
        if (fopen_s(&fp, files[file_idx], "rb") != 0) {
            std::cerr << "can't open " << files[file_idx] << std::endl;
            continue;
        }
        while (n_data < ADJ_MAX_N_DATA) {
            if (fread(&n_discs, 2, 1, fp) < 1)
                break;
            fread(&player, 2, 1, fp);
            fread(host_train_data[n_data].features, 2, ADJ_N_FEATURES, fp);
            fread(&score, 2, 1, fp);
            host_train_data[n_data].score = (float)score * ADJ_STEP;
            if ((n_data & 0xffff) == 0xffff)
                std::cerr << '\r' << n_data;
            score_avg += score;
            ++n_data;
        }
        fclose(fp);
        std::cerr << '\r' << n_data << std::endl;
    }
    score_avg /= n_data;
    std::cerr << std::endl;
    //std::cerr << n_data << " data loaded" << std::endl;
    std::cerr << "score avg " << score_avg << std::endl;
    return n_data;
}

/*
    @brief calculate residual error
*/
__global__ void adj_calculate_residual(const float *device_eval_arr, const int n_data, const int *device_start_idx_arr, const Adj_Data *device_train_data, int *device_rev_idx_arr, float *device_residual_arr, float *device_error_monitor_arr){
    const int data_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (data_idx >= n_data){
        return;
    }
    if (data_idx == 0){
        for (int i = 0; i < N_ERROR_MONITOR; ++i){
            device_error_monitor_arr[i] = 0.0;
        }
    }
    float predicted_value = 0.0;
    for (int i = 0; i < ADJ_N_FEATURES; ++i){
        #if ADJ_CELL_WEIGHT
            if (device_train_data[data_idx].features[i] < 10){
                predicted_value += device_eval_arr[device_train_data[data_idx].features[i]];
            } else if (device_train_data[data_idx].features[i] < 20){
                predicted_value -= device_eval_arr[device_train_data[data_idx].features[i] - 10];
            }
        #else
            predicted_value += device_eval_arr[device_start_idx_arr[i] + (int)device_train_data[data_idx].features[i]];
        #endif
    }
    float residual_error = device_train_data[data_idx].score - predicted_value;
    for (int i = 0; i < ADJ_N_FEATURES; ++i){
        #if ADJ_CELL_WEIGHT
            if (device_train_data[data_idx].features[i] < 10){
                atomicAdd(&device_residual_arr[device_train_data[data_idx].features[i]], residual_error);
                atomicAdd(&device_residual_arr[device_rev_idx_arr[device_train_data[data_idx].features[i]]], residual_error);
            } else if (device_train_data[data_idx].features[i] < 20){
                atomicAdd(&device_residual_arr[device_train_data[data_idx].features[i] - 10], -residual_error);
                atomicAdd(&device_residual_arr[device_rev_idx_arr[device_train_data[data_idx].features[i] - 10]], -residual_error);
            }
        #else
            atomicAdd(&device_residual_arr[device_start_idx_arr[i] + (int)device_train_data[data_idx].features[i]], residual_error);
            int rev_idx = device_rev_idx_arr[device_start_idx_arr[i] + (int)device_train_data[data_idx].features[i]];
            //if (rev_idx != device_start_idx_arr[i] + (int)device_train_data[data_idx].features[i])
            atomicAdd(&device_residual_arr[rev_idx], residual_error);
        #endif
    }
    atomicAdd(&device_error_monitor_arr[0], (residual_error / ADJ_STEP) * (residual_error / ADJ_STEP) / n_data);
    atomicAdd(&device_error_monitor_arr[1], fabs(residual_error / ADJ_STEP) / n_data);
}

/*
    @brief calculate test loss
*/
__global__ void adj_calculate_test_loss(const float *device_eval_arr, const int n_test_data, const int *device_start_idx_arr, const Adj_Data *device_test_data, float *device_test_error_monitor_arr){
    const int data_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (data_idx >= n_test_data){
        return;
    }
    if (data_idx == 0){
        for (int i = 0; i < N_TEST_ERROR_MONITOR; ++i){
            device_test_error_monitor_arr[i] = 0.0;
        }
    }
    float predicted_value = 0.0;
    for (int i = 0; i < ADJ_N_FEATURES; ++i){
        #if ADJ_CELL_WEIGHT
            if (device_test_data[data_idx].features[i] < 10){
                predicted_value += device_eval_arr[device_test_data[data_idx].features[i]];
            } else if (device_test_data[data_idx].features[i] < 20){
                predicted_value -= device_eval_arr[device_test_data[data_idx].features[i] - 10];
            }
        #else
            predicted_value += device_eval_arr[device_start_idx_arr[i] + (int)device_test_data[data_idx].features[i]];
        #endif
    }
    float residual_error = device_test_data[data_idx].score - predicted_value;
    atomicAdd(&device_test_error_monitor_arr[0], (residual_error / ADJ_STEP) * (residual_error / ADJ_STEP) / n_test_data);
    atomicAdd(&device_test_error_monitor_arr[1], fabs(residual_error / ADJ_STEP) / n_test_data);
}

/*
    @brief Gradient Descent Optimizer
*/
__global__ void gradient_descent(const int eval_size, float *device_eval_arr, int *device_n_appear_arr, float *device_residual_arr, float alpha_stab){
    const int eval_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (eval_idx >= eval_size){
        return;
    }
    float lr = alpha_stab / device_n_appear_arr[eval_idx];
    float grad = 2.0 * device_residual_arr[eval_idx];
    if (grad != 0.0){
        device_eval_arr[eval_idx] += lr * grad;
    }
    device_residual_arr[eval_idx] = 0.0;
}

/*
    @brief Momentum Optimizer
*/
__global__ void momentum(const int eval_size, float *device_eval_arr, int *device_n_appear_arr, float *device_residual_arr, float alpha_stab, float *device_m_arr, const int n_loop){
    const int eval_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (eval_idx >= eval_size){
        return;
    }
    float lr = alpha_stab / device_n_appear_arr[eval_idx];
    float grad = 2.0 * device_residual_arr[eval_idx];
    if (grad != 0.0){
        constexpr float beta1 = 0.9;
        device_m_arr[eval_idx] = beta1 * device_m_arr[eval_idx] + lr * grad;
        device_eval_arr[eval_idx] += device_m_arr[eval_idx];
    }
    device_residual_arr[eval_idx] = 0.0;
}

/*
    @brief AdaGrad Optimizer
*/
__global__ void adagrad(const int eval_size, float *device_eval_arr, int *device_n_appear_arr, float *device_residual_arr, float alpha_stab, float *device_v_arr, const int n_loop){
    const int eval_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (eval_idx >= eval_size){
        return;
    }
    float lr = alpha_stab / device_n_appear_arr[eval_idx];
    float grad = 2.0 * device_residual_arr[eval_idx];
    if (grad != 0.0){
        constexpr float beta2 = 0.999;
        constexpr float epsilon = 1e-7;
        device_v_arr[eval_idx] += grad * grad;
        device_eval_arr[eval_idx] += lr * grad / (sqrt(device_v_arr[eval_idx]) + epsilon);
    }
    device_residual_arr[eval_idx] = 0.0;
}

/*
    @brief Adam Optimizer
*/
__global__ void adam(const int eval_size, float *device_eval_arr, int *device_n_appear_arr, float *device_residual_arr, float alpha_stab, float *device_m_arr, float *device_v_arr, const int n_loop){
    const int eval_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (eval_idx >= eval_size){
        return;
    }
    float lr = alpha_stab / device_n_appear_arr[eval_idx];
    float grad = 2.0 * device_residual_arr[eval_idx];
    if (grad != 0.0){
        constexpr float beta1 = 0.9;
        constexpr float beta2 = 0.999;
        constexpr float epsilon = 1e-7;
        float lrt = lr * sqrt(1.0 - pow(beta2, n_loop)) / (1.0 - pow(beta1, n_loop));
        device_m_arr[eval_idx] += (1.0 - beta1) * (grad - device_m_arr[eval_idx]);
        device_v_arr[eval_idx] += (1.0 - beta2) * (grad * grad - device_v_arr[eval_idx]);
        device_eval_arr[eval_idx] += lrt * device_m_arr[eval_idx] / (sqrt(device_v_arr[eval_idx]) + epsilon);
    }
    device_residual_arr[eval_idx] = 0.0;
}

/*
    @brief Output Parameters as integer
*/
void adj_output_param(int eval_size, float *host_eval_arr) {
    for (int i = 0; i < eval_size; ++i) {
        std::cout << (int)round(host_eval_arr[i]) << std::endl;
    }
    std::cerr << "output data fin" << std::endl;
}

/*
    @brief calculate test loss with CPU
*/
void test_loss(float *eval_arr, int *host_start_idx_arr, int n_data, Adj_Data *data, float *mse, float *mae){
    *mse = 0.0;
    *mae = 0.0;
    for (int i = 0; i < n_data; ++i){
        int score = 0;
        for (int j = 0; j < ADJ_N_FEATURES; ++j){
            score += eval_arr[host_start_idx_arr[j] + data[i].features[j]];
        }
        /*
        score += score >= 0 ? ADJ_STEP_2 : -ADJ_STEP_2;
        score /= ADJ_STEP;
        if (score < -SCORE_MAX)
            score = -SCORE_MAX;
        if (score > SCORE_MAX)
            score = SCORE_MAX;
        float abs_error = fabs(data[i].score - score);
        */
        float abs_error = fabs(data[i].score * ADJ_STEP - score) / ADJ_STEP;
        *mse += abs_error * abs_error;
        *mae += abs_error;
    }
    *mse /= n_data;
    *mae /= n_data;
}

int main(int argc, char* argv[]) {
    std::cerr << EVAL_DEFINITION_NAME << std::endl;
    std::cerr << EVAL_DEFINITION_DESCRIPTION << std::endl;
    if (argc < 8) {
        std::cerr << "input [phase] [hour] [minute] [second] [alpha] [n_patience] [in_file] [train_data...]" << std::endl;
        return 1;
    }
    if (argc - 8 >= ADJ_MAX_N_FILES) {
        std::cerr << "too many train files" << std::endl;
        return 1;
    }
    int phase = atoi(argv[1]);
    uint64_t hour = atoi(argv[2]);
    uint64_t minute = atoi(argv[3]);
    uint64_t second = atoi(argv[4]);
    float alpha = atof(argv[5]);
    int n_patience = atoi(argv[6]);
    std::string in_file = (std::string)argv[7];
    char* train_files[ADJ_MAX_N_FILES];
    int n_train_data_file = argc - 8;
    for (int i = 0; i < n_train_data_file; ++i)
        train_files[i] = argv[i + 8];
    second += minute * 60 + hour * 3600;
    uint64_t msecond = second * 1000;

    int eval_size = 0;
    for (int i = 0; i < ADJ_N_EVAL; ++i){
        eval_size += adj_eval_sizes[i];
    }
    std::cerr << "eval_size " << eval_size << std::endl;
    float *host_eval_arr = (float*)malloc(sizeof(float) * eval_size); // eval array
    int *host_rev_idx_arr = (int*)malloc(sizeof(int) * eval_size); // reversed index
    Adj_Data* host_train_data = (Adj_Data*)malloc(sizeof(Adj_Data) * ADJ_MAX_N_DATA); // train data
    int *host_n_appear_arr = (int*)malloc(sizeof(int) * eval_size);
    float *host_error_monitor_arr = (float*)malloc(sizeof(float) * N_ERROR_MONITOR);
    float *host_test_error_monitor_arr = (float*)malloc(sizeof(float) * N_TEST_ERROR_MONITOR);
    if (host_eval_arr == nullptr || host_rev_idx_arr == nullptr || host_train_data == nullptr || host_test_error_monitor_arr == nullptr){
        std::cerr << "cannot allocate memory" << std::endl;
        return 1;
    }
    adj_init_arr(eval_size, host_eval_arr, host_rev_idx_arr, host_n_appear_arr);
    adj_import_eval(in_file, eval_size, host_eval_arr);
    int n_all_data = adj_import_data(n_train_data_file, train_files, host_train_data, host_rev_idx_arr, host_n_appear_arr);
    std::cerr << n_all_data << " data loaded" << std::endl;
    // shuffle data
    std::random_device seed_gen;
    std::mt19937 engine(seed_gen());
    std::shuffle(host_train_data, host_train_data + n_all_data, engine);
    std::cerr << "data shuffled" << std::endl;
    // divide data
    int n_test_data = n_all_data * 0.05; // use 5% as test data
    if (n_test_data <= 0){
        n_test_data = 1;
    }
    int n_train_data = n_all_data - n_test_data;
    Adj_Data* host_test_data = host_train_data + n_train_data;
    std::cerr << "n_train_data " << n_train_data << " n_test_data " << n_test_data << std::endl;
    // calculate n_appear of train data
    int host_start_idx_arr[ADJ_N_FEATURES];
    int start_idx = 0;
    for (int i = 0; i < ADJ_N_FEATURES; ++i){
        if (i > 0){
            if (adj_feature_to_eval_idx[i] > adj_feature_to_eval_idx[i - 1]){
                start_idx += adj_eval_sizes[adj_feature_to_eval_idx[i - 1]];
            }
        }
        host_start_idx_arr[i] = start_idx;
    }
    for (int data_idx = 0; data_idx < n_train_data; ++data_idx){
        for (int i = 0; i < ADJ_N_FEATURES; ++i){
            #if ADJ_CELL_WEIGHT
                if (host_train_data[data_idx].features[i] < 10){
                    ++host_n_appear_arr[host_train_data[data_idx].features[i]];
                    ++host_n_appear_arr[host_rev_idx_arr[host_train_data[data_idx].features[i]]];
                } else if (host_train_data[data_idx].features[i] < 20){
                    ++host_n_appear_arr[host_train_data[data_idx].features[i] - 10];
                    ++host_n_appear_arr[host_rev_idx_arr[host_train_data[data_idx].features[i] - 10]];
                }
            #else
                ++host_n_appear_arr[host_start_idx_arr[i] + (int)host_train_data[data_idx].features[i]];
                int rev_idx = host_rev_idx_arr[host_start_idx_arr[i] + (int)host_train_data[data_idx].features[i]];
                //if (rev_idx != start_idx_arr[i] + (int)host_train_data[data_idx].features[i])
                ++host_n_appear_arr[rev_idx];
            #endif
        }
    }
    for (int i = 0; i < eval_size; ++i){
        host_n_appear_arr[i] = std::min(100, host_n_appear_arr[i]);
    }
    std::cerr << "train data appearance calculated" << std::endl;

    float *device_eval_arr; // device eval array
    int *device_rev_idx_arr; // device reversed index
    Adj_Data *device_train_data;
    Adj_Data *device_test_data;
    int *device_n_appear_arr;
    float *device_residual_arr;
    float *device_error_monitor_arr;
    float *device_test_error_monitor_arr;
    int *device_start_idx_arr;
    hipMalloc(&device_eval_arr, sizeof(float) * eval_size);
    hipMalloc(&device_rev_idx_arr, sizeof(int) * eval_size);
    hipMalloc(&device_train_data, sizeof(Adj_Data) * n_train_data);
    hipMalloc(&device_test_data, sizeof(Adj_Data) * n_test_data);
    hipMalloc(&device_n_appear_arr, sizeof(int) * eval_size);
    hipMalloc(&device_residual_arr, sizeof(float) * eval_size);
    hipMalloc(&device_error_monitor_arr, sizeof(float) * N_ERROR_MONITOR);
    hipMalloc(&device_test_error_monitor_arr, sizeof(float) * N_TEST_ERROR_MONITOR);
    hipMalloc(&device_start_idx_arr, sizeof(int) * ADJ_N_FEATURES);
    hipMemcpy(device_eval_arr, host_eval_arr, sizeof(float) * eval_size, hipMemcpyHostToDevice);
    hipMemcpy(device_rev_idx_arr, host_rev_idx_arr, sizeof(int) * eval_size, hipMemcpyHostToDevice);
    hipMemcpy(device_train_data, host_train_data, sizeof(Adj_Data) * n_train_data, hipMemcpyHostToDevice);
    hipMemcpy(device_test_data, host_test_data, sizeof(Adj_Data) * n_test_data, hipMemcpyHostToDevice);
    hipMemcpy(device_n_appear_arr, host_n_appear_arr, sizeof(int) * eval_size, hipMemcpyHostToDevice);
    hipMemset(device_residual_arr, 0, sizeof(float) * eval_size);
    hipMemcpy(device_start_idx_arr, host_start_idx_arr, sizeof(int) * ADJ_N_FEATURES, hipMemcpyHostToDevice);

    // for adam optimizer
    float *device_m_arr;
    float *device_v_arr;
    hipMalloc(&device_m_arr, sizeof(float) * eval_size);
    hipMalloc(&device_v_arr, sizeof(float) * eval_size);
    hipMemset(device_m_arr, 0, sizeof(float) * eval_size);
    hipMemset(device_v_arr, 0, sizeof(float) * eval_size);
    
    const int n_blocks_test = (n_test_data + N_THREADS_PER_BLOCK_TEST - 1) / N_THREADS_PER_BLOCK_TEST;
    const int n_blocks_residual = (n_train_data + N_THREADS_PER_BLOCK_RESIDUAL - 1) / N_THREADS_PER_BLOCK_RESIDUAL;
    const int n_blocks_next_step = (eval_size + N_THREADS_PER_BLOCK_NEXT_STEP - 1) / N_THREADS_PER_BLOCK_NEXT_STEP;
    std::cerr << "n_blocks_test " << n_blocks_test << " n_blocks_residual " << n_blocks_residual << " n_blocks_next_step " << n_blocks_next_step << std::endl;
    std::cerr << "phase " << phase << std::endl;
    float alpha_stab = alpha; // / n_data;
    adj_calculate_residual <<<n_blocks_residual, N_THREADS_PER_BLOCK_RESIDUAL>>> (device_eval_arr, n_train_data, device_start_idx_arr, device_train_data, device_rev_idx_arr, device_residual_arr, device_error_monitor_arr);
    hipMemcpy(host_error_monitor_arr, device_error_monitor_arr, sizeof(float) * N_ERROR_MONITOR, hipMemcpyDeviceToHost);
    std::cerr << "before MSE " << host_error_monitor_arr[0] << " MAE " << host_error_monitor_arr[1] << std::endl;
    uint64_t strt = tim();
    int n_loop = 0;
    float min_test_mse, min_test_mae;
    int n_test_loss_increase = 0;
    test_loss(host_eval_arr, host_start_idx_arr, n_test_data, host_test_data, &min_test_mse, &min_test_mae);
    while (tim() - strt < msecond){
        ++n_loop;

        // test loss
        adj_calculate_test_loss <<<n_blocks_test, N_THREADS_PER_BLOCK_TEST>>> (device_eval_arr, n_test_data, device_start_idx_arr, device_test_data, device_test_error_monitor_arr);
        hipMemcpy(host_test_error_monitor_arr, device_test_error_monitor_arr, sizeof(float) * N_ERROR_MONITOR, hipMemcpyDeviceToHost);
        if (host_test_error_monitor_arr[0] <= min_test_mse){
            min_test_mse = host_test_error_monitor_arr[0];
            n_test_loss_increase = 0;
        } else{
            ++n_test_loss_increase;
            if (n_test_loss_increase > n_patience){
                break;
            }
        }

        // train loss & residual
        adj_calculate_residual <<<n_blocks_residual, N_THREADS_PER_BLOCK_RESIDUAL>>> (device_eval_arr, n_train_data, device_start_idx_arr, device_train_data, device_rev_idx_arr, device_residual_arr, device_error_monitor_arr);
        hipMemcpy(host_error_monitor_arr, device_error_monitor_arr, sizeof(float) * N_ERROR_MONITOR, hipMemcpyDeviceToHost);

        std::cerr << "\rn_loop " << n_loop << " progress " << (tim() - strt) * 100 / msecond << "% MSE " << host_error_monitor_arr[0] << " MAE " << host_error_monitor_arr[1] << "  test_MSE " << host_test_error_monitor_arr[0] << " test_MAE " << host_test_error_monitor_arr[1] << " loss_inc " << n_test_loss_increase << "                    ";
        
        // next step
        // gradient_descent <<<n_blocks_next_step, N_THREADS_PER_BLOCK_NEXT_STEP>>> (eval_size, device_eval_arr, device_n_appear_arr, device_residual_arr, alpha_stab);
        // momentum <<<n_blocks_next_step, N_THREADS_PER_BLOCK_NEXT_STEP>>> (eval_size, device_eval_arr, device_n_appear_arr, device_residual_arr, alpha_stab, device_m_arr, n_loop);
        // adagrad <<<n_blocks_next_step, N_THREADS_PER_BLOCK_NEXT_STEP>>> (eval_size, device_eval_arr, device_n_appear_arr, device_residual_arr, alpha_stab, device_v_arr, n_loop);
        adam <<<n_blocks_next_step, N_THREADS_PER_BLOCK_NEXT_STEP>>> (eval_size, device_eval_arr, device_n_appear_arr, device_residual_arr, alpha_stab, device_m_arr, device_v_arr, n_loop);
    }
    std::cerr << std::endl;

    // round eval
    hipMemcpy(host_eval_arr, device_eval_arr, sizeof(float) * eval_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < eval_size; ++i){
        host_eval_arr[i] = round(host_eval_arr[i]);
    }
    hipMemcpy(device_eval_arr, host_eval_arr, sizeof(float) * eval_size, hipMemcpyHostToDevice);

    // calculate final loss
    adj_calculate_residual <<<n_blocks_residual, N_THREADS_PER_BLOCK_RESIDUAL>>> (device_eval_arr, n_train_data, device_start_idx_arr, device_train_data, device_rev_idx_arr, device_residual_arr, device_error_monitor_arr);
    hipMemcpy(host_error_monitor_arr, device_error_monitor_arr, sizeof(float) * N_ERROR_MONITOR, hipMemcpyDeviceToHost);
    adj_calculate_test_loss <<<n_blocks_test, N_THREADS_PER_BLOCK_TEST>>> (device_eval_arr, n_test_data, device_start_idx_arr, device_test_data, device_test_error_monitor_arr);
    hipMemcpy(host_test_error_monitor_arr, device_test_error_monitor_arr, sizeof(float) * N_ERROR_MONITOR, hipMemcpyDeviceToHost);
    std::cerr << "phase " << phase << " time " << (tim() - strt) << " ms n_train_data " << n_train_data << " n_test_data " << n_test_data << " n_loop " << n_loop << " MSE " << host_error_monitor_arr[0] << " MAE " << host_error_monitor_arr[1] << " test_MSE " << host_test_error_monitor_arr[0] << " test_MAE " << host_test_error_monitor_arr[1] << " (with int) alpha " << alpha << " n_patience " << n_patience << std::endl;
    std::cout << "phase " << phase << " time " << (tim() - strt) << " ms n_train_data " << n_train_data << " n_test_data " << n_test_data << " n_loop " << n_loop << " MSE " << host_error_monitor_arr[0] << " MAE " << host_error_monitor_arr[1] << " test_MSE " << host_test_error_monitor_arr[0] << " test_MAE " << host_test_error_monitor_arr[1] << " (with int) alpha " << alpha << " n_patience " << n_patience << std::endl;

    // output param
    adj_output_param(eval_size, host_eval_arr);
    return 0;
}
